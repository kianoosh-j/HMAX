#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include "../headers/headers.h"
#include "../headers/picture_handler.h"

#define RAND 10
#define MAX_SIZE 10000
#define ARGN 6
#define GRID 40
#define BLOCK_DIM 128
#define MAX_SHARED 10000


typedef float Picture_Type;



void init_matrix ( Picture &m , int w , int h ){

	if ( w * h >=0 ){
		m.matrix = (Picture_Type *)malloc ( sizeof(Picture_Type) * w * h );
		m.width = w;
		m.height = h;
	}
	else 
		printf ( "error in init_matrix : invalid w and h\n" );
}


void init_gpu_matrix ( Picture &m , int w , int h ){

	if ( w * h >=0 ){
		hipMalloc ( &(m.matrix) , sizeof(Picture_Type) * w * h );
		m.width = w;
		m.height = h;
	}
	else 
		printf (  "error in init_matrix : invalid w and h\n" );
}


void print_matrix ( Picture &m , const char *messeage=NULL ){

	if ( messeage != NULL )
		printf ( "%s\n" , messeage );

	int index = 0;
	for ( int i=0 ; i<m.width ; i++ ){
		for ( int j=0 ; j<m.height ;j++ ){

			printf ( "%10.5f ", m.matrix[index++] );
		}
		printf ( "\n" );
	}
}


void create_random_matrix ( Picture &m , int base ){


	for ( int i=0 ; i<m.width *m.height ; i++ ){

		m.matrix[i] = (Picture_Type)(rand() % base);
	}
}


void __global__ ConvolutionSame ( Picture kernel , Picture main_matrix , Picture conv_matrix ){



	// assign each block to compute some rows of conv matrix
	int conv_height = conv_matrix.height + kernel.height - 1;
	int conv_width = conv_matrix.width + kernel.width - 1;
	int rows_per_block = ceil ( ((float) conv_height ) / gridDim.x );
	int first_row = (blockIdx.x) * rows_per_block;


	// getting shared memory as much as enough
	__shared__ Picture_Type shared_memory [MAX_SHARED];


	// getting first part of shared memory for row of each conv matrix
	Picture_Type *conv_row_shared_memory = &(shared_memory[0]);


	// moving kernel to shared memory
	Picture_Type *kernel_shared = &(shared_memory[conv_width]);
	int temp_index = threadIdx.x;
	int kernel_total_elements = kernel.width * kernel.height;
	while ( temp_index < kernel_total_elements ){
		kernel_shared[temp_index] = kernel.matrix[temp_index];
		temp_index += blockDim.x;
	}
	syncthreads();

	// for each row of conv matrix
	int current_row = first_row + ( (kernel.height - 1) / 2 ) ;
	for ( int i=0    ;     i<rows_per_block && current_row < conv_height     ; i++ , current_row++  ){

		// find rang rows of kernel and main matrix that should multiply to each other to do this row of conv matrix
		int kernel_top_bound = (kernel.height -1) - max (kernel.height - current_row -1 , 0 );
		int kernel_bot_bound = (kernel.height -1) - min ( kernel.height-1 , conv_height - current_row - 1  );
		int main_matrix_top_bound = max ( current_row + 1 - kernel.height , 0 );
		int main_matrix_bot_bound = min ( current_row , main_matrix.height -1 );


		// reset the shared memory for this row
		int temp_index = threadIdx.x;
		while ( temp_index < conv_width ){
			conv_row_shared_memory[temp_index] = 0;
			temp_index += blockDim.x;
		}
		syncthreads();


		// for each row in range
		for ( 		int main_row = main_matrix_top_bound , kernel_row = kernel_top_bound 	;	
				main_row <= main_matrix_bot_bound && kernel_row >= kernel_bot_bound	;
				main_row++ , kernel_row--
		    ){

			// set pointers for walking over rows
			Picture_Type *main_row_pointer = &( main_matrix.matrix[ main_row*main_matrix.width ] ) + threadIdx.x; // it points to the first of row + thread Index
			Picture_Type *kernel_row_pointer = &( kernel_shared [ kernel_row * kernel.width ] ); // it points to the begining of row in kernel
			Picture_Type *conv_row_pointer = conv_row_shared_memory + threadIdx.x;

			// mult all elements of both rows of kernel and main matrix
			int main_col = threadIdx.x;
			while ( true ){

				// check if is finished using the thread 0 main_col
				if ( (main_col - threadIdx.x) >= main_matrix.width ){
					break;
				}

				// multiply this col to all columns of kernel row
				int kernel_col = 0;
				Picture_Type main_element_temp = *main_row_pointer;
				while ( kernel_col < kernel.width ){

					if ( main_col < main_matrix.width ){
						*conv_row_pointer += main_element_temp * (*kernel_row_pointer);
					}
					syncthreads();
					kernel_col++;
					kernel_row_pointer++;
					conv_row_pointer++;
				}

				// increment main_col and correct pointers
				main_col += blockDim.x;
				main_row_pointer += blockDim.x;
				kernel_row_pointer += kernel.width;
				conv_row_pointer += ( blockDim.x - kernel.width );
			}
		}
		// end for


		// move this shared memory to the conv matrix in global memory
		temp_index = threadIdx.x + (  kernel.width / 2 );
		Picture_Type *conv_row_pointer = & ( conv_matrix.matrix [ (current_row-1) * conv_matrix.width ] );
		int thread_index = threadIdx.x;
		while ( thread_index < conv_matrix.width  ){

			conv_row_pointer [ thread_index ] = conv_row_shared_memory [ temp_index ];
			temp_index += blockDim.x;
			thread_index += blockDim.x;
		}
		// end moving

	}
	// end for
}


void __global__ Convolution ( Picture kernel , Picture main_matrix , Picture conv_matrix ){



	// assign each block to compute some rows of conv matrix
	//int conv_height = conv_matrix.height + kernel.height - 1;
	//int conv_width = conv_matrix.width + kernel.width - 1;
	int rows_per_block = ceil ( ((float) conv_matrix.height ) / gridDim.x );
	int first_row = (blockIdx.x) * rows_per_block;


	// getting shared memory as much as enough
	__shared__ Picture_Type shared_memory [MAX_SHARED];


	// getting first part of shared memory for row of each conv matrix
	Picture_Type *conv_row_shared_memory = &(shared_memory[0]);


	// moving kernel to shared memory
	Picture_Type *kernel_shared = &(shared_memory[conv_matrix.width]);
	int temp_index = threadIdx.x;
	int kernel_total_elements = kernel.width * kernel.height;
	while ( temp_index < kernel_total_elements ){
		kernel_shared[temp_index] = kernel.matrix[temp_index];
		temp_index += blockDim.x;
	}
	syncthreads();

	// for each row of conv matrix
	int current_row = first_row;
	for ( int i=0    ;     i<rows_per_block && current_row < conv_matrix.height     ; i++ , current_row++  ){

		//if ( threadIdx.x == 0)
		//	printf ( "%d %d\n" , blockIdx.x , current_row );
		// find rang rows of kernel and main matrix that should multiply to each other to do this row of conv matrix
		int kernel_top_bound = (kernel.height -1) - max (kernel.height - current_row -1 , 0 );
		int kernel_bot_bound = (kernel.height -1) - min ( kernel.height-1 , conv_matrix.height - current_row - 1  );
		int main_matrix_top_bound = max ( current_row + 1 - kernel.height , 0 );
		int main_matrix_bot_bound = min ( current_row , main_matrix.height -1 );


		// reset the shared memory for this row
		int temp_index = threadIdx.x;
		while ( temp_index < conv_matrix.width ){
			conv_row_shared_memory[temp_index] = 0;
			temp_index += blockDim.x;
		}
		syncthreads();


		// for each row in range
		for ( 		int main_row = main_matrix_top_bound , kernel_row = kernel_top_bound 	;	
				main_row <= main_matrix_bot_bound && kernel_row >= kernel_bot_bound	;
				main_row++ , kernel_row--
		    ){

			// set pointers for walking over rows
			Picture_Type *main_row_pointer = &( main_matrix.matrix[ main_row*main_matrix.width ] ) + threadIdx.x; // it points to the first of row + thread Index
			Picture_Type *kernel_row_pointer = &( kernel_shared [ kernel_row * kernel.width ] ); // it points to the begining of row in kernel
			Picture_Type *conv_row_pointer = conv_row_shared_memory + threadIdx.x;

			// mult all elements of both rows of kernel and main matrix
			int main_col = threadIdx.x;
			while ( true ){

				// check if is finished using the thread 0 main_col
				if ( (main_col - threadIdx.x) >= main_matrix.width ){
					break;
				}

				// multiply this col to all columns of kernel row
				int kernel_col = 0;
				Picture_Type main_element_temp = *main_row_pointer;
				while ( kernel_col < kernel.width ){

					if ( main_col < main_matrix.width ){
						*conv_row_pointer += main_element_temp * (*kernel_row_pointer);
					}
					syncthreads();
					kernel_col++;
					kernel_row_pointer++;
					conv_row_pointer++;
				}

				// increment main_col and correct pointers
				main_col += blockDim.x;
				main_row_pointer += blockDim.x;
				kernel_row_pointer += kernel.width;
				conv_row_pointer += ( blockDim.x - kernel.width );
			}
		}
		// end for


		// move this shared memory to the conv matrix in global memory
		temp_index = threadIdx.x;
		Picture_Type *conv_row_pointer = & ( conv_matrix.matrix [ current_row * conv_matrix.width ] );
		while ( temp_index < conv_matrix.width  ){

			conv_row_pointer [ temp_index ] = conv_row_shared_memory [ temp_index ];
			temp_index += blockDim.x;
		}
		// end moving

	}
	// end for
}


void __global__ FilterSame ( Picture kernel , Picture main_matrix , Picture conv_matrix ){


	int conv_height = conv_matrix.height + kernel.height - 1;
	int conv_width = conv_matrix.width + kernel.width - 1;
	// assign each block to compute some rows of conv matrix
	int rows_per_block = ceil ( ((float) conv_height ) / gridDim.x );
	int first_row = blockIdx.x * rows_per_block;


	// getting shared memory as much as enough
	__shared__ Picture_Type shared_memory [MAX_SHARED];


	// getting first part of shared memory for row of each conv matrix
	Picture_Type *conv_row_shared_memory = &(shared_memory[0]);


	// moving kernel to shared memory
	Picture_Type *kernel_shared = &(shared_memory[conv_width]);
	int temp_index = threadIdx.x;
	int kernel_total_elements = kernel.width * kernel.height;
	while ( temp_index < kernel_total_elements ){
		kernel_shared[temp_index] = kernel.matrix[temp_index];
		temp_index += blockDim.x;
	}
	syncthreads();

	// for each row of conv matrix
	int current_row = first_row + ( (kernel.height - 1) / 2 ) ;
	for ( int i=0    ;     i<rows_per_block && current_row < conv_height     ; i++ , current_row++  ){

		// find rang rows of kernel and main matrix that should multiply to each other to do this row of conv matrix
		int kernel_top_bound = max (kernel.height - current_row -1 , 0 );
		int kernel_bot_bound = min ( kernel.height-1 , conv_height - current_row - 1  );
		int main_matrix_top_bound = max ( current_row + 1 - kernel.height , 0 );
		int main_matrix_bot_bound = min ( current_row , main_matrix.height -1 );


		// reset the shared memory for this row
		int temp_index = threadIdx.x;
		while ( temp_index < conv_width ){
			conv_row_shared_memory[temp_index] = 0;
			temp_index += blockDim.x;
		}
		syncthreads();

		// for each row in range
		for ( 		int main_row = main_matrix_top_bound , kernel_row = kernel_top_bound 	;	
				main_row <= main_matrix_bot_bound && kernel_row <= kernel_bot_bound	;
				main_row++ , kernel_row++ 
		    ){

			// set pointers for walking over rows
			Picture_Type *main_row_pointer = &( main_matrix.matrix[ main_row*main_matrix.width ] ) + threadIdx.x; // it points to the first of row + thread Index
			Picture_Type *kernel_row_pointer = &( kernel_shared [ (kernel_row+1)*kernel.width -1 ] ); // it points to the end of row in kernel
			Picture_Type *conv_row_pointer = conv_row_shared_memory + threadIdx.x;

			// mult all elements of both rows of kernel and main matrix
			int main_col = threadIdx.x;
			while ( true ){

				// check if is finished using the thread 0 main_col
				if ( (main_col - threadIdx.x) >= main_matrix.width ){
					break;
				}

				// multiply this col to all columns of kernel row
				int kernel_col = kernel.width-1;
				Picture_Type main_element_temp = *main_row_pointer;
				while ( kernel_col >= 0 ){

					if ( main_col < main_matrix.width ){
						*conv_row_pointer += main_element_temp * (*kernel_row_pointer);
					}
					syncthreads();
					kernel_col--;
					kernel_row_pointer--;
					conv_row_pointer++;
				}

				// increment main_col and correct pointers
				main_col += blockDim.x;
				main_row_pointer += blockDim.x;
				kernel_row_pointer += kernel.width;
				conv_row_pointer += blockDim.x - kernel.width;
			}

		}
		// end for

		// move this shared memory to the conv matrix in global memory
		temp_index = threadIdx.x + (  kernel.width / 2 );
		Picture_Type *conv_row_pointer = & ( conv_matrix.matrix [ (current_row-1) * conv_matrix.width ] );
		int thread_index = threadIdx.x;
		while ( thread_index < conv_matrix.width  ){

			conv_row_pointer [ thread_index ] = conv_row_shared_memory [ temp_index ];
			temp_index += blockDim.x;
			thread_index += blockDim.x;
		}






		// end moving

	}
	// end for
}


void __global__ Filter ( Picture kernel , Picture main_matrix , Picture conv_matrix ){


	// assign each block to compute some rows of conv matrix
	int rows_per_block = ceil ( ((float) conv_matrix.height ) / gridDim.x );
	int first_row = blockIdx.x * rows_per_block;


	// getting shared memory as much as enough
	__shared__ Picture_Type shared_memory [MAX_SHARED];


	// getting first part of shared memory for row of each conv matrix
	Picture_Type *conv_row_shared_memory = &(shared_memory[0]);


	// moving kernel to shared memory
	Picture_Type *kernel_shared = &(shared_memory[conv_matrix.width]);
	int temp_index = threadIdx.x;
	int kernel_total_elements = kernel.width * kernel.height;
	while ( temp_index < kernel_total_elements ){
		kernel_shared[temp_index] = kernel.matrix[temp_index];
		temp_index += blockDim.x;
	}
	syncthreads();

	// for each row of conv matrix
	int current_row = first_row ;
	for ( int i=0    ;     i<rows_per_block && current_row < conv_matrix.height     ; i++ , current_row++  ){

		// find rang rows of kernel and main matrix that should multiply to each other to do this row of conv matrix
		int kernel_top_bound = max (kernel.height - current_row -1 , 0 );
		int kernel_bot_bound = min ( kernel.height-1 , conv_matrix.height - current_row - 1  );
		int main_matrix_top_bound = max ( current_row + 1 - kernel.height , 0 );
		int main_matrix_bot_bound = min ( current_row , main_matrix.height -1 );


		// reset the shared memory for this row
		int temp_index = threadIdx.x;
		while ( temp_index < conv_matrix.width ){
			conv_row_shared_memory[temp_index] = 0;
			temp_index += blockDim.x;
		}
		syncthreads();


		// for each row in range
		for ( 		int main_row = main_matrix_top_bound , kernel_row = kernel_top_bound 	;	
				main_row <= main_matrix_bot_bound && kernel_row <= kernel_bot_bound	;
				main_row++ , kernel_row++ 
				){

			// set pointers for walking over rows
			Picture_Type *main_row_pointer = &( main_matrix.matrix[ main_row*main_matrix.width ] ) + threadIdx.x; // it points to the first of row + thread Index
			Picture_Type *kernel_row_pointer = &( kernel_shared [ (kernel_row+1)*kernel.width -1 ] ); // it points to the end of row in kernel
			Picture_Type *conv_row_pointer = conv_row_shared_memory + threadIdx.x;

			// mult all elements of both rows of kernel and main matrix
			int main_col = threadIdx.x;
			while ( true ){

				// check if is finished using the thread 0 main_col
				if ( (main_col - threadIdx.x) >= main_matrix.width ){
					break;
				}

				// multiply this col to all columns of kernel row
				int kernel_col = kernel.width-1;
				Picture_Type main_element_temp = *main_row_pointer;
				while ( kernel_col >= 0 ){

					if ( main_col < main_matrix.width ){
						*conv_row_pointer += main_element_temp * (*kernel_row_pointer);
					}
					syncthreads();
					kernel_col--;
					kernel_row_pointer--;
					conv_row_pointer++;
				}

				// increment main_col and correct pointers
				main_col += blockDim.x;
				main_row_pointer += blockDim.x;
				kernel_row_pointer += kernel.width;
				conv_row_pointer += blockDim.x - kernel.width;
			}

		}
		// end for


		// move this shared memory to the conv matrix in global memory
		temp_index = threadIdx.x;
		Picture_Type *conv_row_pointer = & ( conv_matrix.matrix [ current_row * conv_matrix.width ] );
		while ( temp_index < conv_matrix.width ){

			conv_row_pointer [ temp_index ] = conv_row_shared_memory [ temp_index ];
			temp_index += blockDim.x;
		}
		// end moving

	}
	// end for
}


int imfilter_pictures_same_gpu ( Picture &p1 , Picture &p2 , Picture &pres ){

	if ( !valid_picture (p2) || !equal_size_pictures (p1,pres) ){
		cerr<<"error : conv2_pictures_same : size of pictures not OK\n";
		return -1;
	}

	Picture dev_p1,dev_p2,dev_pres;

	dev_p1.width = p1.width;
	dev_p1.height = p1.height;

	dev_p2.width = p1.width;
	dev_p2.height = p2.height;

	dev_pres.width = pres.width;
	dev_pres.height = pres.height;


	hipMalloc((void**)&dev_p1.matrix,sizeof(float)*p1.width*p1.height );
	hipMalloc((void**)&dev_p2.matrix,sizeof(float)*p2.width*p2.height );
	hipMalloc((void**)&dev_pres.matrix,sizeof(float)*pres.width*pres.height );
	
	hipMemcpy(dev_p1.matrix,p1.matrix,p1.width*p1.height*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(dev_p2.matrix,p2.matrix,p2.width*p2.height*sizeof(float),hipMemcpyHostToDevice);

	//FilterSame<<<p1.height/4, p1.width/4>>> (dev_p2,dev_p1,dev_pres);

	hipMemcpy(pres.matrix,dev_pres.matrix,p1.width*p1.height*sizeof(float),hipMemcpyDeviceToHost);

	hipFree (dev_p1.matrix );
	hipFree (dev_p2.matrix );
	hipFree (dev_pres.matrix);

	return 0;
}


int conv2_pictures_same_gpu ( Picture &p1 , Picture &p2 , Picture &pres ){


	if ( !valid_picture ( p2 ) || !equal_size_pictures ( p1 , pres ) ){
		cerr<<"error : conv2_pictures_same : size of pictures not OK\n";
		return -1;
	}

	Picture dev_p1,dev_p2,dev_pres;

	dev_p1.width = p1.width;
	dev_p1.height = p1.height;

	dev_p2.width = p1.width;
	dev_p2.height = p2.height;

	dev_pres.width = pres.width;
	dev_pres.height = pres.height;


	hipMalloc((void**)&dev_p1.matrix,sizeof(float)*p1.width*p1.height );
	hipMalloc((void**)&dev_p2.matrix,sizeof(float)*p2.width*p2.height );
	hipMalloc((void**)&dev_pres.matrix,sizeof(float)*pres.width*pres.height );

	hipMemcpy(dev_p1.matrix,p1.matrix,p1.width*p1.height*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(dev_p2.matrix,p2.matrix,p2.width*p2.height*sizeof(float),hipMemcpyHostToDevice);

	//ConvolutionSame<<<p1.height/4,p1.width/4>>> ( dev_p2 , dev_p1 , dev_pres ); 

	hipMemcpy(pres.matrix,dev_pres.matrix,pres.width*pres.height*sizeof(float),hipMemcpyDeviceToHost);


	hipFree (dev_p1.matrix );
	hipFree (dev_p2.matrix );
	hipFree (dev_pres.matrix);

	return 0;

}




